
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>

__device__ static inline void setSeed(int64_t *seed)
{
    *seed = (*seed ^ 0x5deece66d) & ((1LL << 48) - 1);
}

__device__ static inline int next(int64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1LL << 48) - 1);
    return (int) (*seed >> (48 - bits));
}

__device__ static inline int nextInt(int64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if((m & n) == 0) return (int) ((n * (int64_t)next(seed, 31)) >> 31);

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

struct Pos
{
    int x, z;
};

__device__  class BoundingBox {
public:
	Pos start;
	Pos end;
	__device__ static BoundingBox getBoundingBox(int minx, int miny, int minz, int maxx, int maxy, int maxz) {
		BoundingBox box;
		box.start.x = minx;
		box.start.z = minz;
		box.end.x = maxx;
		box.end.z = maxz;
		return box;
	}
	__device__ bool intersectsWith(BoundingBox box)
	{
		return this->end.x >= box.start.x && this->start.x <= box.end.x && this->end.z >= box.start.z && this->start.z <= box.end.z;
	}
};

#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}
FILE *fp;   
uint64_t total = 0;
uint64_t current = 0;

__device__ BoundingBox guessBox;

uint64_t* buffer;
uint32_t* counter;
//__device__ uint64_t hardcoded = 8682522807148012UL * 181783497276652981UL;

__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(uint64_t offset, uint32_t* counter, uint64_t* buffer){
    uint64_t seed = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    //int64_t structureSeed = hardcoded ^ seed;
    int64_t structureSeed = seed;
    BoundingBox spawnBox;
    Pos spawn;
    spawn.x = 0;
    spawn.z = 0;
    int count = 0;
    setSeed(&structureSeed);
    nextInt(&structureSeed, 12000);
    for(spawn.z = 0; !spawnBox.intersectsWith(guessBox) && count <= 150; spawn.z += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64))
    {
        spawn.x += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64);
        spawnBox.start = spawn;
        spawnBox.end = spawn;
        count++;
    }
    if(spawnBox.intersectsWith(guessBox)){
        buffer[atomicAdd(counter, 1)] = seed;
    }
}
__global__ __launch_bounds__(1,1) static void setupGuessBox(Pos guessMin, Pos guessMax){
    guessBox.start = guessMin;
    guessBox.end = guessMax;
}
int main(int argc, char **argv ){
    time_t start = time(NULL);
    fp = fopen("seananners.txt", "w+");
    double seconds_per_structure_seed = 0.0;
    std::vector<std::thread> threads;
    std::cout << "Begin loading threads" << std::endl;  
    int thread = 0;
    int curr = 0;
    uint64_t startValue = 0;
    total = 100000000000;
    std::vector<std::string> tArr;
    int tmpCount = 0;
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(uint64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    Pos guessMin;
    Pos guessMax;
    guessMin.x = 1710;
    guessMin.z = 276;
    guessMax.x = 1734;
    guessMax.z = 348;
    setupGuessBox<<<1,1>>>(guessMin, guessMax);
    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());  
    std::vector<uint64_t> results;
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(uint64_t offset = startValue; offset <= total; offset += SEEDS_PER_CALL){
            threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(offset, counter, buffer);
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());  
            for(int i = 0; i < *counter; i++){
                uint64_t seed = buffer[i];
                if(seed != 0)
                    fprintf(fp, "%lld\n", seed);
            }
            *counter = 0;
            if(countOut >= 5000000000){
                time_t tempTime = time(NULL);
                uint64_t tempDiff = tempTime - start;
                double sps = (double)(offset - startValue)/tempDiff;
                double percent = ((double)offset/(double)total) * 100.0;
                printf("Seeds Per Second: %f\tProgress: %f\n", sps, percent);
                countOut = 0;
            }
        countOut += SEEDS_PER_CALL;
    }

    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}