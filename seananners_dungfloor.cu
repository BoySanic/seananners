
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>

#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}
__device__ static  int next(int64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1LL << 48) - 1);
    return (int) (*seed >> (48 - bits));
}

__device__ static  int nextInt(int64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if((m & n) == 0) return (int) ((n * (int64_t)next(seed, 31)) >> 31);

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}
__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(int64_t offset, uint32_t* counter, int64_t* buffer){
    uint64_t worldSeed = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    int64_t seed = worldSeed;
    int64_t tempSeed = (seed * 21586261248413UL + 164331561754775UL) & 281474976710655UL; 
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
 
    tempSeed = (tempSeed * 25214903917UL + 11UL) & 281474976710655UL;
    tempSeed = (tempSeed * 25214903917UL + 11UL) & 281474976710655UL;
    tempSeed = (tempSeed * 25214903917UL + 11UL) & 281474976710655UL;

    int xWiggle = nextInt(&tempSeed, 2) + 2;
    int zWiggle = nextInt(&tempSeed, 2) + 2;
    if(!(xWiggle == 3 && zWiggle == 2))return;

    //0th
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //first column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //second column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //third column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) != 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //fourth column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) != 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //fifth column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) != 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //sixth column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) != 0)return;
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    //seventh column
    seed = (seed * 25214903917UL + 11UL) & 281474976710655UL;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) == 0)return;
    if(next(&seed, 2) != 0)return;
    if(next(&seed, 2) == 0)return;
    buffer[atomicAdd(counter, 1)] = worldSeed;
}

int64_t* buffer;
uint32_t* counter;

int main(int argc, char **argv ){
    int64_t startValue  = 0;
    int64_t total       = 281474976710656; 
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-dfloor.txt", "w+");
    double seconds_per_structure_seed = 0.0;
    int thread = 0;
    int curr = 0;

    uint64_t amount = total - startValue;
    int tmpCount = 0;
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(int64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());

    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(int64_t offset = 0; offset < amount; offset += SEEDS_PER_CALL){
        int64_t value = startValue + offset;
        threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(value, counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        for(int i = 0; i < *counter; i++){
            int64_t timeGuess = buffer[i];
            fprintf(fp, "%lld\n", timeGuess);
        }
        if(countOut >= 100000000000){
            time_t tempTime = time(NULL);
            uint64_t tempDiff = tempTime - start;
            double sps = (double)offset/(double)tempDiff;
            double percent = ((double)offset/(double)amount) * 100.0;
            printf("Seeds Per Second: %f\tProgress: %f\n", sps, percent);
            countOut = 0;
        }
        *counter = 0;
        countOut += SEEDS_PER_CALL;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}