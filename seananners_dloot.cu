#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>
__device__ static inline void setSeed(int64_t *seed)
{
    *seed = (*seed ^ 0x5deece66d) & ((1LL << 48) - 1);
}

__device__ static inline int next(int64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1LL << 48) - 1);
    return (int) (*seed >> (48 - bits));
}

__device__ static inline int nextInt(int64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if((m & n) == 0) return (int) ((n * (int64_t)next(seed, 31)) >> 31);

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}
__device__ static inline void skip65(int64_t *seed){
    *seed = (*seed * 0xB4500F159B6D  + 0x5593A16ED14B ) & ((1LL << 48) - 1);
}
__device__ static inline void skip63(int64_t *seed){
    *seed = (*seed * 0x89A36E758065 + 0xD75D8F3C9E9 ) & ((1LL << 48) - 1);
}



#define BLOCK_SIZE (256)
#define WORK_SIZE_BITS 20
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}


enum Item{
    unset,
    saddle,
    ingotIron,
    bread,
    wheat,
    gunpowder,
    silk,
    bucketEmpty,
    appleGold,
    redstone,
    record
};
struct Pos
{
    int x, z;
};

struct ItemStack{
    Item id;
    int amount;
};
const __device__ ItemStack Chest1[27] = {
    {},
    {},
    {},
    {},
    {},
    {gunpowder, 1}, //5
    {}, //6
    {}, //7 
    {}, //8
    {}, //9
    {}, //10
    {}, //11
    {}, //12
    {}, //13
    {}, //14
    {}, //15
    {}, //16
    {}, //17
    {}, //18
    {}, //19
    {}, //20
    {silk, 1}, //21
    {gunpowder, 1}, //22
    {}, //23
    {}, //24
    {}, //25
    {saddle, 1}, //26
};
const __device__ ItemStack Chest2[27] = {
    {},
    {},
    {},
    {},
    {},
    {bucketEmpty, 1}, //5
    {}, //6
    {gunpowder, 1}, //7
    {saddle, 1}, //8
    {}, //9
    {}, //10
    {}, //11
    {}, //12
    {}, //13
    {wheat, 1}, //14
    {}, //15
    {}, //16
    {}, //17
    {gunpowder, 3}, //18
};
//__device__ ItemStack *Chest1;
//__device__ ItemStack *Chest2;
__device__ static void getItem(ItemStack* tempItem, int64_t* seed){

    int x = nextInt(seed, 11);
    if(x == 0){
        tempItem->id = saddle;
        tempItem->amount = 1;
    }
    if(x == 1){
        tempItem->id = ingotIron;
        tempItem->amount = nextInt(seed, 4) + 1;
    }
    if(x == 2){
        tempItem->id = bread;
        tempItem->amount = 1;
    }
    if(x == 3){
        tempItem->id = wheat;
        tempItem->amount = nextInt(seed, 4) + 1;
    }
    if(x == 4){
        tempItem->id = gunpowder;
        tempItem->amount = nextInt(seed, 4) + 1;
    }
    if(x == 5){
        tempItem->id = silk;
        tempItem->amount = nextInt(seed, 4) + 1;
    }
    if(x == 6){
        tempItem->id = bucketEmpty;
        tempItem->amount = 1;
    }
    if(x == 7 && nextInt(seed, 100) == 0){
        tempItem->id = appleGold;
    }
    if(x == 8 && nextInt(seed, 2) == 0){
        tempItem->id = redstone;
        tempItem->amount = nextInt(seed, 4) + 1;
    }
    if(x == 9 && nextInt(seed, 10) == 0){
        tempItem->id = record;
        tempItem->amount = 1;
        //We don't have one anyway so I'm unconcerned
    }
    if(x > 9){
        tempItem->id = unset;
        tempItem->amount = 1;
    }

}
__device__ static bool testSeed(int64_t seed){
    int64_t testSeed = seed;
    int64_t permutationSeed = testSeed;
    int chestCounter = 0;
    int curChest = 0;
    int itemCounter = 0;
    int firstChest = 0;
    ItemStack firstChestSim[27];
    ItemStack secondChestSim[27];
    for(int i = 0; i < 2; i++){
        for(int i2 = 0; i2 < 3; i2++){
            int curChest = 0;
            int locCounter = 0;
            /*
                int xChest = (x + random.nextInt(xWiggle * 2 + 1)) - xWiggle;
                int yChest = ySpawner;
                int zChest = (z + random.nextInt(zWiggle * 2 + 1)) - zWiggle;
            */
            int xChest = (nextInt(&permutationSeed, 7));
            int zChest = (nextInt(&permutationSeed, 5));
            if(xChest == 6 && zChest == 2)
                curChest = 2;
            else if(xChest == 4 && zChest == 4)
                curChest = 1;
            if(curChest == firstChest){
                return false;
            }
            if((xChest == 6 && zChest == 2) || xChest == 4 && zChest == 4){
                for(int i3 = 0; i3 <= 8; i3++){
                    ItemStack it;
                    getItem(&it, &permutationSeed);
                    if(it.id == saddle || it.id == gunpowder || it.id == wheat || it.id == bucketEmpty || it.id == silk){
                        int itemIndex = nextInt(&permutationSeed, 26);
                        if (Chest1[itemIndex].id == it.id && Chest1[itemIndex].amount > firstChestSim[itemIndex].amount && && firstChestSim[itemIndex].amount + it.amount <= Chest1[itemIndex].amount && curChest == 1){
                            firstChestSim[itemIndex].id = it.id;
                            firstChestsim[itemIndex].amount += it.amount;
                        }
                        if (Chest2[itemIndex].id == it.id && Chest2[itemIndex].amount > secondChestSim[itemIndex].amount && && secondChestSim[itemIndex].amount + it.amount <= Chest2[itemIndex].amount && curChest == 2){
                            secondChestSim[itemIndex].id = it.id;
                            secondChestSim[itemIndex].amount += it.amount;
                        }
                    }
                    else{
                        return false;
                    }
                }
                if(locCounter > 4 && curChest == 1 || locCounter > 5 && curChest == 2){
                    return false;
                }
                itemCounter += locCounter;
            }
            else{
                locCounter = 0;
            }
            if(firstChest == 0){
                firstChest = curChest;
            }
        }
    }
    if(itemCounter == 9){
        return true;
    }
    else{
        return false;
    }
}
__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(int64_t offset, uint32_t* counter, int64_t* buffer){
    uint64_t seed = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    int64_t structureSeed = seed;
    int count = 0;
    setSeed(&structureSeed);
    nextInt(&structureSeed, 16);
    nextInt(&structureSeed, 128);
    nextInt(&structureSeed, 16);
    int xWiggle = nextInt(&structureSeed, 2) + 2;
    int zWiggle = nextInt(&structureSeed, 2) + 2;
    if(xWiggle == 3 && zWiggle == 2){
        skip63(&structureSeed);
        if(testSeed(structureSeed)){
            buffer[atomicAdd(counter, 1)] = seed;
        }
    }
}
__device__ int64_t stonks[] = {3033227586,
    11299383782,
    19174124756,
    26213759191,
    30882125013,
    31573082574};
__global__ __launch_bounds__(1,1) static void testFunc(){
    for(int64_t i = 0; i < 6; i++){
        int64_t structureSeed = stonks[i];
        skip63(&structureSeed);
        if(testSeed(structureSeed)){
            printf("THIS SEED IS STONKS: %lld\n", stonks[i]);
        }
        else{
            printf("THIS SEED IS NOT STONKS: %lld\n", stonks[i]);
        }
    }
}

/*__global__ __launch_bounds__(1,1) static void setupChestTables(){
    Chest1 = (ItemStack*) malloc(sizeof(ItemStack) * 27);
    Chest2 = (ItemStack*) malloc(sizeof(ItemStack) * 27);

    Chest1[5].id = gunpowder;
    Chest1[5].amount = 1;
    Chest1[21].id = silk;
    Chest1[21].amount = 1;
    Chest1[22].id = gunpowder;
    Chest1[22].amount = 1;
    Chest1[26].id = saddle;
    Chest1[26].amount = 1;

    Chest2[5].id = bucketEmpty;
    Chest2[5].amount = 1;
    Chest2[7].id = gunpowder;
    Chest2[7].amount = 2;
    Chest2[8].id = saddle;
    Chest2[8].amount = 1;
    Chest2[14].id = wheat;
    Chest2[14].amount = 1;
    Chest2[18].id = gunpowder;
    Chest2[18].amount = 3;
}*/
int64_t* buffer;
uint32_t* counter;

std::vector<int64_t> structureSeeds;
int64_t* structSeedsArr;
int main(int argc, char **argv ){
    int64_t startValue  = 0;
    int64_t total       = 281474976710656; 
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-dloot.txt", "w+");
    double seconds_per_structure_seed = 0.0;
    int thread = 0;
    int curr = 0;

    uint64_t amount = total - startValue;
    int tmpCount = 0;
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(int64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());

    //setupChestTables<<<1,1>>>();
    //printf("Chest tables set up\n");

    //testFunc<<<1,1>>>();
    for(int i = 0; i < 27; i++){
        printf("%d, %d index: %d\n", Chest1[i].id, Chest1[i].amount, i);
    }
    for(int i = 0; i < 27; i++){
        printf("%d, %d index: %d\n", Chest2[i].id, Chest2[i].amount, i);
    }
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(int64_t offset = 0; offset < amount; offset += SEEDS_PER_CALL){
        int64_t value = startValue + offset;
        threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(value, counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        for(int i = 0; i < *counter; i++){
            int64_t timeGuess = buffer[i];
            fprintf(fp, "%lld\n", timeGuess);
        }
        if(countOut >= 20000000000){
            time_t tempTime = time(NULL);
            uint64_t tempDiff = tempTime - start;
            double sps = (double)offset/(double)tempDiff;
            double percent = ((double)offset/(double)amount) * 100.0;
            printf("Seeds Per Second: %f\tProgress: %f\n", sps, percent);
            countOut = 0;
        }
        *counter = 0;
        countOut += SEEDS_PER_CALL;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}