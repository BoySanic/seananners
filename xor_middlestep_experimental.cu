
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>

#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}
/*__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(int64_t* values, int size, uint64_t offset, uint32_t* counter, uint64_t* buffer){
    int64_t Time = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    int64_t scrambledTime = hardcoded ^ Time;
    if(binarySearch(values, scrambledTime, 0, size) != -1){
        buffer[atomicAdd(counter, 1)] = Time;
        return;
    }
}*/
/*
Originally 64-bit seed value.
Mod 48 bit to get the 48 bit value.
Time could be any 64-bit value that when mod 48 gives the structure seed value.
We have the 48 bit post-mod 48 value
((8682522807148012UL * 181783497276652981UL)^x)%(1LL << 48) = someSeed


Take 48 bit seed value
Loop upper bits
Xor (8682522807148012UL * 181783497276652981UL) with upperBits Seed
Find seed that matches
*/
__device__ uint64_t hardcoded = 8682522807148012UL * 181783497276652981UL;
__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(uint64_t baseValue, uint32_t* counter, uint64_t* buffer){
    int upperBits = (blockIdx.x * blockDim.x + threadIdx.x);
    if(upperBits > 65535){
        //printf("upperbits: %d", upperBits);
        return;
    }
    printf("%d\n", upperBits);
    int64_t seed = ( upperBits << 48) | baseValue;
    int64_t timeGuess = seed ^ hardcoded;
    //printf("test: %lld\n", timeGuess);
    if(((timeGuess ^ hardcoded) % (1LL << 48)) != baseValue){
        printf("What the frick\n");
    }
    if(timeGuess <= 1282780799000000000 && timeGuess >= 1280361600000000000)
        buffer[atomicAdd(counter, 1)] = timeGuess;
    //else
        //printf("Value: %lld\n", timeGuess);
    //if(timeGuess <= 2505600000000000)
      //  buffer[atomicAdd(counter, 1)] = timeGuess;
}
uint64_t* buffer;
uint32_t* counter;
std::vector<int64_t> structureSeeds;
int64_t* structSeedsArr;
int main(int argc, char **argv ){
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-middlestep.txt", "w+");
    std::fstream infile;
    infile.open("seananners.txt", std::ios::in);
    std::string line;
    while(std::getline(infile, line)){
        int64_t structureSeed = 0;
        std::istringstream iss(line);
        if(!(iss >> structureSeed)){break;}
        structureSeeds.push_back(structureSeed);
    }
    infile.close();
    double seconds_per_structure_seed = 0.0;
    std::vector<std::thread> threads;
    int thread = 0;
    int curr = 0;
    uint64_t startValue = 0;
    uint64_t total = 281474976710656;
    int tmpCount = 0;
    int tmpSize = structureSeeds.size();
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(uint64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&structSeedsArr, sizeof(uint64_t) * tmpSize));
    GPU_ASSERT(hipPeekAtLastError());

    for(int i = 0; i <= structureSeeds.size(); i++){
        structSeedsArr[i] = structureSeeds[i];
    }

    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(uint64_t offset = 0; offset < tmpSize; offset++){
        uint64_t inputValue = structSeedsArr[offset];
        threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(inputValue, counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        for(int i = 0; i < *counter; i++){
            int64_t timeGuess = buffer[i];
            if(timeGuess != 0)
                fprintf(fp, "%lld\n", timeGuess);
        }
        *counter = 0;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}