
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>

#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}
__device__ uint64_t hardcoded = 8682522807148012UL * 181783497276652981UL;
__device__ int binarySearch(int64_t* values, int64_t value, int start, int end){
    int low = 0;
    int high = end - 1;
    int mid = 0;
    if(high < value || low > value)
        return -1;
    while(low <= high){
        mid = low + ((high - low)/2);
        if(values[mid] > value) 
            high = mid - 1;
        else if((values[mid] < value))
            low = mid - 1;
        else
            return mid;
    }
    return -1;
}
/*__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(int64_t* values, int size, uint64_t offset, uint32_t* counter, uint64_t* buffer){
    int64_t Time = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    int64_t scrambledTime = hardcoded ^ Time;
    if(binarySearch(values, scrambledTime, 0, size) != -1){
        buffer[atomicAdd(counter, 1)] = Time;
        return;
    }
}*/
__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(int64_t* values, int size, uint64_t offset, uint32_t* counter, uint64_t* buffer){
    int64_t Time = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    int64_t scrambledTime = hardcoded ^ Time;
    if(binarySearch(values, scrambledTime, 0, size) != -1){
        buffer[atomicAdd(counter, 1)] = Time;
        return;
    }
}
uint64_t* buffer;
uint32_t* counter;
std::vector<int64_t> structureSeeds;
int64_t* structSeedsArr;
int main(int argc, char **argv ){
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-middlestep.txt", "w+");
    std::fstream infile;
    infile.open("seananners.txt", std::ios::in);
    std::string line;
    while(std::getline(infile, line)){
        int64_t structureSeed = 0;
        std::istringstream iss(line);
        if(!(iss >> structureSeed)){break;}
        structureSeeds.push_back(structureSeed);
    }
    infile.close();
    double seconds_per_structure_seed = 0.0;
    std::vector<std::thread> threads;
    int thread = 0;
    int curr = 0;
    uint64_t startValue = 0;
    uint64_t total = 281474976710656;
    int tmpCount = 0;
    int tmpSize = structureSeeds.size();
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(uint64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&structSeedsArr, sizeof(uint64_t) * tmpSize));
    GPU_ASSERT(hipPeekAtLastError());
    printf("test1\n");
    for(int i = 0; i <= structureSeeds.size(); i++){
        structSeedsArr[i] = structureSeeds[i];
    }
    printf("test2\n");

    printf("test3\n");
    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    uint64_t countOut = 0;
    uint64_t tempCount;
    printf("test4\n");
    for(uint64_t offset = startValue; offset <= total; offset += SEEDS_PER_CALL){
            threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(structSeedsArr, tmpSize, offset, counter, buffer);
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());  
            for(int i = 0; i < *counter; i++){
                uint64_t seed = buffer[i];
                if(seed != 0)
                    fprintf(fp, "%lld\n", seed);
            }
            *counter = 0;
            if(countOut >= 100000000000){
                time_t tempTime = time(NULL);
                uint64_t tempDiff = tempTime - start;
                uint64_t sps = (uint64_t)(offset - startValue)/tempDiff;
                double percent = ((double)offset/(double)total) * 100.0;
                printf("Seeds Per Second: %lld\tProgress: %f\n", sps, percent);
                countOut = 0;
            }
        countOut += SEEDS_PER_CALL;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}