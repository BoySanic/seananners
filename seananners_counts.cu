
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>
__device__ static inline void setSeed(int64_t *seed)
{
    *seed = (*seed ^ 0x5deece66d) & ((1LL << 48) - 1);
}

__device__ static inline int next(int64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1LL << 48) - 1);
    return (int) (*seed >> (48 - bits));
}

__device__ static inline int nextInt(int64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if((m & n) == 0) return (int) ((n * (int64_t)next(seed, 31)) >> 31);

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

struct Pos
{
    int x, z;
};

__device__  class BoundingBox {
public:
	Pos start;
	Pos end;
	__device__ static BoundingBox getBoundingBox(int minx, int miny, int minz, int maxx, int maxy, int maxz) {
		BoundingBox box;
		box.start.x = minx;
		box.start.z = minz;
		box.end.x = maxx;
		box.end.z = maxz;
		return box;
	}
	__device__ bool intersectsWith(BoundingBox box)
	{
		return this->end.x >= box.start.x && this->start.x <= box.end.x && this->end.z >= box.start.z && this->start.z <= box.end.z;
	}
};
#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}

/*
Originally 64-bit seed value.
Mod 48 bit to get the 48 bit value.
Time could be any 64-bit value that when mod 48 gives the structure seed value.
We have the 48 bit post-mod 48 value
((8682522807148012UL * 181783497276652981UL)^x)%(1LL << 48) = someSeed


Take 48 bit seed value
Loop upper bits
Xor (8682522807148012UL * 181783497276652981UL) with upperBits Seed
Find seed that matches
*/
__device__ BoundingBox guessBox;
__device__ int64_t hardcoded = 8682522807148012L * 181783497276652981L;
typedef unsigned long long int uint64_cu;
__device__ static bool match(int64_t seed){
    BoundingBox spawnBox;
    Pos spawn;
    spawn.x = 0;
    spawn.z = 0;
    int count = 0;
    int64_t structureSeed = seed;
    setSeed(&structureSeed);
    nextInt(&structureSeed, 12000);
    for(spawn.z = 0; (!spawnBox.intersectsWith(guessBox) && count <= 150) && !(spawn.z >= guessBox.end.z || spawn.x >= guessBox.end.x); spawn.z += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64))
    {
        spawn.x += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64);
        spawnBox.start = spawn;
        spawnBox.end = spawn;
        count++;
    }
    if(spawnBox.intersectsWith(guessBox)){
        return true;
    }
    return false;
}
__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(uint64_t offset, uint64_cu* underCounter, uint64_cu* overCounter, int64_t* buffer){
    int64_t timeGuess = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    uint64_t seedIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    int64_t seedGuess = hardcoded ^ timeGuess;
    int64_t structureSeed = seedGuess;
    BoundingBox spawnBox;
    Pos spawn;
    spawn.x = 0;
    spawn.z = 0;
    int count = 0;
    setSeed(&structureSeed);
    nextInt(&structureSeed, 12000);
    for(spawn.z = 0; (!spawnBox.intersectsWith(guessBox) && count <= 150); spawn.z += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64))
    {
        spawn.x += nextInt(&structureSeed, 64) - nextInt(&structureSeed, 64);
        spawnBox.start = spawn;
        spawnBox.end = spawn;
        count++;
    }
    if(spawn.z > guessBox.end.z || spawn.x > guessBox.end.x){
        atomicAdd(underCounter, 1);
        return;
    }
    if(spawn.z < guessBox.start.z || spawn.x < guessBox.start.x){
        atomicAdd(overCounter, 1);
        return;
    }
}
__global__ __launch_bounds__(1,1) static void setupGuessBox(Pos guessMin, Pos guessMax){
    guessBox.start = guessMin;
    guessBox.end = guessMax;
}
int64_t* buffer;
uint32_t* counter;
uint64_cu* underCounter;
uint64_cu* overCounter;
int main(int argc, char **argv ){
    int64_t startValue  = 1282521600000;
    int64_t total       = 1282780799000;
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-middlestep.txt", "w+");
    double seconds_per_structure_seed = 0.0;
    int thread = 0;
    int curr = 0;

    uint64_t amount = total - startValue;
    int tmpCount = 0;
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(int64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    GPU_ASSERT(hipMallocManaged(&overCounter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    GPU_ASSERT(hipMallocManaged(&underCounter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    Pos guessMin;
    Pos guessMax;
    guessMin.x = 1710;
    guessMin.z = 276;
    guessMax.x = 1734;
    guessMax.z = 348;
    setupGuessBox<<<1,1>>>(guessMin, guessMax);
    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    //*counter = 0;
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(int64_t offset = 0; offset < amount; offset += SEEDS_PER_CALL){
        int64_t value = startValue + offset;
        value *= 1000;
        threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(value, underCounter, overCounter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        /*for(int i = 0; i < *counter; i++){
            int64_t timeGuess = buffer[i];
            if(timeGuess == -1){
                *underCounter++;
            }
            if(timeGuess == 1){
                *overCounter++;
            }
        }*/
        if(countOut >= 1000000000){
            time_t tempTime = time(NULL);
            uint64_t tempDiff = tempTime - start;
            double sps = (double)offset/(double)tempDiff;
            double percent = ((double)offset/(double)amount) * 100.0;
            printf("Seeds Per Second: %f\tProgress: %f\n", sps, percent);
            countOut = 0;
        }
        //*counter = 0;
        countOut += SEEDS_PER_CALL;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    uint64_t tot = total - startValue;
    printf("Time taken: %lld\nSeeds per second: %15.9f\nUnderCounter: %lld\nOverCounter: %lld\nTotal: %lld", diff, seedsPerSec, *underCounter, *overCounter, tot);
    fclose(fp);
    return 0;
}