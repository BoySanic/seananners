
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <thread>
#include <ctime>
#include <stdio.h>
__device__ static inline void setSeed(int64_t *seed)
{
    *seed = (*seed ^ 0x5deece66d) & ((1LL << 48) - 1);
}

__device__ static inline int next(int64_t *seed, const int bits)
{
    *seed = (*seed * 0x5deece66d + 0xb) & ((1LL << 48) - 1);
    return (int) (*seed >> (48 - bits));
}

__device__ static inline int nextInt(int64_t *seed, const int n)
{
    int bits, val;
    const int m = n - 1;

    if((m & n) == 0) return (int) ((n * (int64_t)next(seed, 31)) >> 31);

    do {
        bits = next(seed, 31);
        val = bits % n;
    }
    while (bits - val + m < 0);
    return val;
}

struct Pos
{
    int x, z;
};

__device__  class BoundingBox {
public:
	Pos start;
	Pos end;
	__device__ bool intersectsWith(BoundingBox box)
	{
		return this->end.x >= box.start.x && this->start.x <= box.end.x && this->end.z >= box.start.z && this->start.z <= box.end.z;
	}
};
#define BLOCK_SIZE (128)
#define WORK_SIZE_BITS 16
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}

/*
Originally 64-bit seed value.
Mod 48 bit to get the 48 bit value.
Time could be any 64-bit value that when mod 48 gives the structure seed value.
We have the 48 bit post-mod 48 value
((8682522807148012UL * 181783497276652981UL)^x)%(1LL << 48) = someSeed


Take 48 bit seed value
Loop upper bits
Xor (8682522807148012UL * 181783497276652981UL) with upperBits Seed
Find seed that matches
*/
__device__ BoundingBox guessBox;
__device__ int64_t startCurrent = 8682522807148012L;
__device__ int64_t hardcoded = 181783497276652981L;
__device__ int64_t current;

__global__ __launch_bounds__(BLOCK_SIZE,2) static void threadWork(uint64_t offset, uint32_t* counter, int64_t* buffer){
    int64_t timeGuess = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
    timeGuess *= 1000;
    int64_t seedGuess = current ^ timeGuess;
    nextInt(&seedGuess, 203);
    if(nextInt(&seedGuess, 203) == 103){
        buffer[atomicAdd(counter, 1)] = timeGuess;
    }
}
__global__ __launch_bounds__(1,1) static void setupGuessBox(Pos guessMin, Pos guessMax){
    current = startCurrent*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded*hardcoded;
    guessBox.start = guessMin;
    guessBox.end = guessMax;
}
int64_t* buffer;
uint32_t* counter;
std::vector<int64_t> structureSeeds;
int64_t* structSeedsArr;

int main(int argc, char **argv ){
    int64_t startValue  = 1282613228000000;
    int64_t total       = 1282706397225000;
    time_t start = time(NULL);
    FILE* fp = fopen("seananners-middlestep.txt", "w+");
    double seconds_per_structure_seed = 0.0;
    int thread = 0;
    int curr = 0;

    uint64_t amount = total - startValue;
    int tmpCount = 0;
    GPU_ASSERT(hipMallocManaged(&buffer, sizeof(int64_t) * SEEDS_PER_CALL));
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipMallocManaged(&counter, sizeof(uint32_t)));
    GPU_ASSERT(hipPeekAtLastError());
    Pos guessMin;
    Pos guessMax;
    guessMin.x = 1710;
    guessMin.z = 276;
    guessMax.x = 1734;
    guessMax.z = 348;
    setupGuessBox<<<1,1>>>(guessMin, guessMax);
    hipSetDevice(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    uint64_t countOut = 0;
    uint64_t tempCount;
    for(int64_t offset = 0; offset < amount; offset += SEEDS_PER_CALL){

        if(offset > amount){
            break;
        }
        int64_t value = startValue + offset;
        threadWork<<<1ULL<<WORK_SIZE_BITS,BLOCK_SIZE>>>(value, counter, buffer);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());  
        for(int i = 0; i < *counter; i++){
            int64_t timeGuess = buffer[i];
            fprintf(fp, "%lld\n", timeGuess);
        }
        if(countOut >= 1000000000){
            time_t tempTime = time(NULL);
            uint64_t tempDiff = tempTime - start;
            double sps = (double)offset/(double)tempDiff;
            double percent = ((double)offset/(double)amount) * 100.0;
            printf("Seeds Per Second: %f\tProgress: %f\n", sps, percent);
            countOut = 0;
        }
        *counter = 0;
        countOut += SEEDS_PER_CALL;
    }
    time_t end = time(NULL);
    uint64_t diff = end - start;
    double seedsPerSec = (double)total/(double)diff;
    printf("Time taken: %lld\nSeeds per second: %15.9f", diff, seedsPerSec);
    fclose(fp);
    return 0;
}